#include "hip/hip_runtime.h"
# include "conv.h"

// Construction function of convolution layer.
ConvolutionLayer :: ConvolutionLayer(int _in_channels, int _out_channels, int _size_r, int _size_c, int _kernel_size_r, int _kernel_size_c, int _stride_r, int _stride_c, int _padding_r, int _padding_c) {
    in_channels = _in_channels;
    out_channels = _out_channels;
    kernel_size_r = _kernel_size_r;
    kernel_size_c = _kernel_size_c;
    stride_r = _stride_r;
    stride_c = _stride_c;
    padding_r = _padding_r;
    padding_c = _padding_c;
    size_r = _size_r;
    size_c = _size_c;
    out_size_r = (size_r - kernel_size_r + 2 * padding_r) / stride_r + 1;
    out_size_c = (size_c - kernel_size_c + 2 * padding_c) / stride_c + 1;
    
    channel_N = in_channels * out_channels;
    kernel_N = kernel_size_r * kernel_size_c;
    output_N = out_channels * out_size_r * out_size_c;
    input_N = in_channels * size_r * size_c;
    total_N = kernel_N * channel_N;

    hipMalloc((void **)&weight, sizeof(double) * channel_N * kernel_N);
    hipMalloc((void **)&bias, sizeof(double) * out_channels);
    set_params();
}

// Destruction function of convolution layer.
ConvolutionLayer :: ~ConvolutionLayer() {
    hipFree(weight);
    hipFree(bias);
    if (h_weight != NULL) free(h_weight);
    if (h_bias != NULL) free(h_bias);
}

// Convolution forward (cpu)
void conv_forward_cpu(double* input, double *output, double *weight, double *bias, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int output_N = out_channels * out_size_r * out_size_c;
    const int input_N = in_channels * size_r * size_c;
    for (int b = 0; b < batch_size; ++ b) {
        for (int out_ch = 0; out_ch < out_channels; ++ out_ch)
            for (int r = 0; r < out_size_r; ++ r)
                for (int c = 0; c < out_size_c; ++ c) {
                    output[b * output_N + (out_ch * out_size_r + r) * out_size_c + c] = bias[out_ch];
                    for (int kr = 0; kr < kernel_size_r; ++ kr) {
                        for (int kc = 0; kc < kernel_size_c; ++ kc) {
                            const int input_r = r * stride_r + kr - padding_r;
                            const int input_c = c * stride_c + kc - padding_c;
                            if (input_r >= 0 && input_r < size_r && input_c >= 0 && input_c < size_c) {
                                for (int in_ch = 0; in_ch < in_channels; ++ in_ch) {
                                    output[b * output_N + (out_ch * out_size_r + r) * out_size_c + c] += weight[((out_ch * in_channels + in_ch) * kernel_size_r + kr) * kernel_size_c + kc] * input[b * input_N + (in_ch * size_r + input_r) * size_c + input_c];
                                }
                            }
                        }
                    }
                }
    }
}

double* ConvolutionLayer :: cpu_forward(double *input, const int batch_size) {
    double *output;
    output = (double *) malloc (sizeof(double) * batch_size * output_N);
    conv_forward_cpu(input, output, h_weight, h_bias, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    return output;
}

// Convolution forward (weight) (basic, no optimization)
__global__ void conv_forward_basic_weight(double *input, double *output, double *weight, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int output_N = out_channels * out_size_r * out_size_c;
    const int input_N = in_channels * size_r * size_c;
    const long long total_N = 1ll * kernel_size_r * kernel_size_c * out_size_r * out_size_c * in_channels * out_channels;
    const long long begin_idx = total_N * thread_pos / total_threads;
    const long long end_idx = total_N * (thread_pos + 1) / total_threads;
    for (long long i = begin_idx; i < end_idx; ++ i) {
        long long temp = i;
        const int i_kernel_c = temp % kernel_size_c;
        const int i_kernel_r = (temp /= kernel_size_c) % kernel_size_r;
        const int i_in_channel = (temp /= kernel_size_r) % in_channels;
        const int i_out_channel = (temp /= in_channels) % out_channels;
        const int i_out_c = (temp /= out_channels) % out_size_c;
        const int i_out_r = (temp /= out_size_c) % out_size_r;
        const int input_c = i_out_c * stride_c + i_kernel_c - padding_c;
        const int input_r = i_out_r * stride_r + i_kernel_r - padding_r;
        const int i_channel = i_out_channel * in_channels + i_in_channel;
        if (input_r >= 0 && input_r < size_r && input_c >= 0 && input_c < size_c)
            atomicAdd(
                &output[batch_id * output_N + (i_out_channel * out_size_r + i_out_r) * out_size_c + i_out_c], 
                weight[(i_channel * kernel_size_r + i_kernel_r) * kernel_size_c + i_kernel_c] * input[batch_id * input_N + (i_in_channel * size_r + input_r) * size_c + input_c]
            );
    }
}

// Convolution forward (bias) (basic, no optimization)
__global__ void conv_forward_basic_bias(double *input, double *output, double *bias, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int output_N = out_channels * out_size_r * out_size_c;
    const int begin_idx = output_N * thread_pos / total_threads;
    const int end_idx = output_N * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++ i) {
        int temp = i;
        const int i_channel = temp % out_channels;
        const int i_out_c = (temp /= out_channels) % out_size_c;
        const int i_out_r = (temp /= out_size_c) % out_size_r;
        output[batch_id * output_N + (i_channel * out_size_r + i_out_r) * out_size_c + i_out_c] = input[batch_id * output_N + (i_channel * out_size_r + i_out_r) * out_size_c + i_out_c] + bias[i_channel];
    }
}

double* ConvolutionLayer :: basic_forward(dim3 grid, dim3 block, double *input, const int batch_size) {
    double *output;
    hipMalloc((void **)&output, sizeof(double) * batch_size * output_N);
    hipMemset(output, 0, sizeof(double) * batch_size * output_N);
    conv_forward_basic_weight <<<grid, block>>> (input, output, weight, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    conv_forward_basic_bias <<<grid, block>>> (output, output, bias, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    hipDeviceSynchronize();
    return output;
}

void ConvolutionLayer :: set_params(double *_h_weight, double *_h_bias) {
    if (_h_weight == NULL) {
        h_weight = (double*) malloc (sizeof(double) * channel_N * kernel_N);
        for (int i = 0; i < channel_N * kernel_N; ++ i)
            h_weight[i] = init_rand();
    } else {
        if (h_weight != NULL) free(h_weight);
        h_weight = _h_weight;
    }
    if (_h_bias == NULL) {
        h_bias = (double*) malloc (sizeof(double) * out_channels);
        for (int i = 0; i < out_channels; ++ i)
            h_bias[i] = init_rand();
    } else{
        if (h_bias != NULL) free(h_bias);
        h_bias = _h_bias;
    }
    hipMemcpy(weight, h_weight, sizeof(double) * channel_N * kernel_N, hipMemcpyHostToDevice);
    hipMemcpy(bias, h_bias, sizeof(double) * out_channels, hipMemcpyHostToDevice);
}

void ConvolutionLayer :: get_output_size(int &output_r, int &output_c) {
    output_r = out_size_r;
    output_c = out_size_c;
}

/*
void conv_cudnn_forward(
    hipdnnHandle_t& handle,
    double *input,
    double *output,
    double *weight,
    double *bias,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int size_r,
    const int size_c,
    const int kernel_size_r,
    const int kernel_size_c,
    const int stride_r,
    const int stride_c,
    const int padding_r,
    const int padding_c
) {
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            in_channels,
            size_r,
            size_c
        )
    );

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(
        hipdnnSetFilter4dDescriptor(
            kernel_descriptor,
            HIPDNN_DATA_DOUBLE,
            HIPDNN_TENSOR_NCHW,
            out_channels,
            in_channels,
            kernel_size_r,
            kernel_size_c
        )
    );

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(
        hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            padding_r,
            padding_c,
            stride_r,
            stride_c,
            1,
            1,
            HIPDNN_CROSS_CORRELATION,
            HIPDNN_DATA_DOUBLE
        )
    );

    const int out_size_r = (size_r - kernel_size_r + 2 * padding_r) / stride_r + 1;
    const int out_size_c = (size_c - kernel_size_c + 2 * padding_c) / stride_c + 1;

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            out_channels,
            out_size_r,
            out_size_c
        )
    );

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    hipdnnConvolutionFwdAlgoPerf_t convolution_algorithm_perf[4];
    int returned_cnt;
    checkCUDNN(
        cudnnGetConvolutionForwardAlgorithm_v7(
            handle,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            4,
            &returned_cnt,
            convolution_algorithm_perf
        )
    );

    bool found_algo = false;
    for (int n = 0; n < returned_cnt; ++ n) {
        if (convolution_algorithm_perf[n].status == HIPDNN_STATUS_SUCCESS && 
            convolution_algorithm_perf[n].algo != HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED) {
            convolution_algorithm = convolution_algorithm_perf[n].algo;
            found_algo = true;
            break;
        }
    }
    if (! found_algo) {
        std :: cerr << "No convolution algorithm is found." << std :: endl;
        std :: exit(EXIT_FAILURE);
    }

    size_t workspace_bytes;
    checkCUDNN(
        hipdnnGetConvolutionForwardWorkspaceSize(
            handle,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            convolution_algorithm,
            &workspace_bytes
        )
    );


    if (workspace_bytes == 0) workspace_bytes = (size_t)(1 << 23);
    void *workspace = NULL;
    hipMalloc(&workspace, workspace_bytes);

    const double alpha = 1.0, beta = 0.0;
    const size_t output_bytes = sizeof(double) * batch_size * out_channels * out_size_r * out_size_c;

    checkCUDNN(
        hipdnnConvolutionForward(
            handle,
            &alpha,
            input_descriptor,
            input,
            kernel_descriptor,
            weight,
            convolution_descriptor,
            convolution_algorithm,
            workspace,
            output_bytes,
            &beta,
            output_descriptor,
            output
        )
    );

    if (bias != NULL) {
        hipdnnTensorDescriptor_t bias_descriptor;
        checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCUDNN(
            hipdnnSetTensor4dDescriptor(
                bias_descriptor,
                HIPDNN_TENSOR_NCHW,
                HIPDNN_DATA_DOUBLE,
                1,
                out_channels,
                1,
                1
            )
        );
        checkCUDNN(
            hipdnnAddTensor(
                handle,
                &alpha,
                bias_descriptor,
                bias,
                &beta,
                output_descriptor,
                output
            )
        );
        hipdnnDestroyTensorDescriptor(bias_descriptor);
    }

    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
}

double* ConvolutionLayer :: cudnn_forward(hipdnnHandle_t &handle, double *input, const int batch_size) {    
    double *output;
    hipMalloc((void **)&output, sizeof(double) * batch_size * output_N);
    hipMemset(output, 0, sizeof(double) * batch_size * output_N);
    conv_cudnn_forward(handle, input, output, weight, bias, batch_size, in_channels, out_channels, size_r, size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    return output;
}
*/