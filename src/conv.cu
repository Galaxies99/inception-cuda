#include "hip/hip_runtime.h"
# include "conv.h"

// Construction function of convolution layer.
ConvolutionLayer :: ConvolutionLayer(int _in_channels, int _out_channels, int _size_r, int _size_c, int _kernel_size_r, int _kernel_size_c, int _stride_r, int _stride_c, int _padding_r, int _padding_c) {
    in_channels = _in_channels;
    out_channels = _out_channels;
    kernel_size_r = _kernel_size_r;
    kernel_size_c = _kernel_size_c;
    stride_r = _stride_r;
    stride_c = _stride_c;
    padding_r = _padding_r;
    padding_c = _padding_c;
    size_r = _size_r;
    size_c = _size_c;
    out_size_r = (size_r - kernel_size_r + 2 * padding_r) / stride_r + 1;
    out_size_c = (size_c - kernel_size_c + 2 * padding_c) / stride_c + 1;
    
    channel_N = in_channels * out_channels;
    kernel_N = kernel_size_r * kernel_size_c * out_size_r * out_size_c;
    output_N = out_channels * out_size_r * out_size_c;
    input_N = in_channels * size_r * size_c;
    total_N = kernel_N * channel_N;

    hipMalloc((void **)&weight, sizeof(float) * channel_N * kernel_N);
    hipMalloc((void **)&bias, sizeof(float) * out_channels);
    set_params();
}

// Destruction function of convolution layer.
ConvolutionLayer :: ~ConvolutionLayer() {
    hipFree(weight);
    hipFree(bias);
    if (h_weight != NULL) free(h_weight);
    if (h_bias != NULL) free(h_bias);
}

// Convolution forward (cpu)
void conv_forward_cpu(float* input, float *output, float *weight, float *bias, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int output_N = out_channels * out_size_r * out_size_c;
    const int input_N = in_channels * size_r * size_c;
    for (int b = 0; b < batch_size; ++ b) {
        for (int out_ch = 0; out_ch < out_channels; ++ out_ch)
            for (int r = 0; r < out_size_r; ++ r)
                for (int c = 0; c < out_size_c; ++ c) {
                    output[b * output_N + (out_ch * out_size_r + r) * out_size_c + c] = bias[out_ch];
                    for (int kr = 0; kr < kernel_size_r; ++ kr) {
                        for (int kc = 0; kc < kernel_size_c; ++ kc) {
                            const int input_r = r * stride_r + kr - padding_r;
                            const int input_c = c * stride_c + kc - padding_c;
                            if (input_r >= 0 && input_r < size_r && input_c >= 0 && input_c < size_c) {
                                for (int in_ch = 0; in_ch < in_channels; ++ in_ch) {
                                    output[b * output_N + (out_ch * out_size_r + r) * out_size_c + c] += weight[((out_ch * in_channels + in_ch) * kernel_size_r + kr) * kernel_size_c + kc] * input[b * input_N + (in_ch * size_r + input_r) * size_c + input_c];
                                }
                            }
                        }
                    }
                }
    }
}

float* ConvolutionLayer :: cpu_forward(float *input, const int batch_size) {
    float *output;
    output = (float *) malloc (sizeof(float) * batch_size * output_N);
    conv_forward_cpu(input, output, h_weight, h_bias, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    return output;
}

// Convolution forward (weight) (basic, no optimization)
__global__ void conv_forward_basic_weight(float *input, float *output, float *weight, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int output_N = out_channels * out_size_r * out_size_c;
    const int input_N = in_channels * size_r * size_c;
    const int total_N = kernel_size_r * kernel_size_c * out_size_r * out_size_c * in_channels * out_channels;
    const int begin_idx = total_N * thread_pos / total_threads;
    const int end_idx = total_N * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++ i) {
        int temp = i;
        const int i_kernel_c = temp % kernel_size_c;
        const int i_kernel_r = (temp /= kernel_size_c) % kernel_size_r;
        const int i_in_channel = (temp /= kernel_size_r) % in_channels;
        const int i_out_channel = (temp /= in_channels) % out_channels;
        const int i_out_c = (temp /= out_channels) % out_size_c;
        const int i_out_r = (temp /= out_size_c) % out_size_r;
        const int input_c = i_out_c * stride_c + i_kernel_c - padding_c;
        const int input_r = i_out_r * stride_r + i_kernel_r - padding_r;
        const int i_channel = i_out_channel * in_channels + i_in_channel;
        if (input_r >= 0 && input_r < size_r && input_c >= 0 && input_c < size_c)
            atomicAdd(
                &output[batch_id * output_N + (i_out_channel * out_size_r + i_out_r) * out_size_c + i_out_c], 
                weight[(i_channel * kernel_size_r + i_kernel_r) * kernel_size_c + i_kernel_c] * input[batch_id * input_N + (i_in_channel * size_r + input_r) * size_c + input_c]
            );
    }
}

// Convolution forward (bias) (basic, no optimization)
__global__ void conv_forward_basic_bias(float *input, float *output, float *bias, const int batch_size, const int in_channels, const int out_channels, const int size_r, const int size_c, const int out_size_r, const int out_size_c, const int kernel_size_r, const int kernel_size_c, const int stride_r, const int stride_c, const int padding_r, const int padding_c) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int output_N = out_channels * out_size_r * out_size_c;
    const int begin_idx = output_N * thread_pos / total_threads;
    const int end_idx = output_N * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++ i) {
        int temp = i;
        const int i_channel = temp % out_channels;
        const int i_out_c = (temp /= out_channels) % out_size_c;
        const int i_out_r = (temp /= out_size_c) % out_size_r;
        output[batch_id * output_N + (i_channel * out_size_r + i_out_r) * out_size_c + i_out_c] = input[batch_id * output_N + (i_channel * out_size_r + i_out_r) * out_size_c + i_out_c] + bias[i_channel];
    }
}

float* ConvolutionLayer :: basic_forward(dim3 grid, dim3 block, float *input, const int batch_size) {
    float *output;
    hipMalloc((void **)&output, sizeof(float) * batch_size * output_N);
    hipMemset(output, 0, sizeof(float) * batch_size * output_N);
    conv_forward_basic_weight <<<grid, block>>> (input, output, weight, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    conv_forward_basic_bias <<<grid, block>>> (output, output, bias, batch_size, in_channels, out_channels, size_r, size_c, out_size_r, out_size_c, kernel_size_r, kernel_size_c, stride_r, stride_c, padding_r, padding_c);
    hipDeviceSynchronize();
    return output;
}

void ConvolutionLayer :: set_params(float *_h_weight, float *_h_bias) {
    if (_h_weight == NULL) {
        h_weight = (float*) malloc (sizeof(float) * channel_N * kernel_N);
        for (int i = 0; i < channel_N * kernel_N; ++ i)
            h_weight[i] = init_rand();
    } else {
        if (h_weight != NULL) free(h_weight);
        h_weight = _h_weight;
    }
    if (_h_bias == NULL) {
        h_bias = (float*) malloc (sizeof(float) * out_channels);
        for (int i = 0; i < out_channels; ++ i)
            h_bias[i] = init_rand();
    } else{
        if (h_bias != NULL) free(h_bias);
        h_bias = _h_bias;
    }
    hipMemcpy(weight, h_weight, sizeof(float) * channel_N * kernel_N, hipMemcpyHostToDevice);
    hipMemcpy(bias, h_bias, sizeof(float) * out_channels, hipMemcpyHostToDevice);
}

void ConvolutionLayer :: get_output_size(int &output_r, int &output_c) {
    output_r = out_size_r;
    output_c = out_size_c;
}
