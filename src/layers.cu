# include "layers.h"


InceptionLayer5 :: InceptionLayer5(const int in_channels, const int size) : in_channels(in_channels), size(size), way1_1(in_channels, 192, size, size), way1_2(192, 320, size, size, 3, 3, 2, 2, 0, 0), way2_1(in_channels, 192, size, size), way2_2(192, 192, size, size, 1, 7, 1, 1, 0, 3), way2_3(192, 192, size, size, 7, 1, 1, 1, 3, 0), way2_4(192, 192, size, size, 3, 3, 2, 2, 0, 0), maxpool(in_channels, size, 3, 2) {
    out_size = (size - 3) / 2 + 1;
    out_channels = 320 + 192 + 768;
}

int InceptionLayer5 :: get_out_size() const {
    return out_size;
}

int InceptionLayer5 :: get_out_channels() const {
    return out_channels;
}

void InceptionLayer5 :: set_params(struct InceptionLayer5params params) {
    way1_1.set_params(params.way1_1_w, params.way1_1_b);
    way1_2.set_params(params.way1_2_w, params.way1_2_b);
    way2_1.set_params(params.way2_1_w, params.way2_1_b);
    way2_2.set_params(params.way2_2_w, params.way2_2_b);
    way2_3.set_params(params.way2_3_w, params.way2_3_b);
    way2_4.set_params(params.way2_4_w, params.way2_4_b);
}

double* InceptionLayer5 :: cpu_forward(double *input, const int batch_size) {
    // way1
    double *way1_o1 = way1_1.cpu_forward(input, batch_size);
    cpu_relu(way1_o1, batch_size * 192 * size * size);
    double *way1_o = way1_2.cpu_forward(way1_o1, batch_size);
    cpu_relu(way1_o, batch_size * 320 * out_size * out_size);
    free(way1_o1);
    // way2
    double *way2_o1 = way2_1.cpu_forward(input, batch_size);
    cpu_relu(way2_o1, batch_size * 192 * size * size);
    double *way2_o2 = way2_2.cpu_forward(way2_o1, batch_size);
    cpu_relu(way2_o2, batch_size * 192 * size * size);
    double *way2_o3 = way2_3.cpu_forward(way2_o2, batch_size);
    cpu_relu(way2_o3, batch_size * 192 * size * size);
    double *way2_o = way2_4.cpu_forward(way2_o3, batch_size);
    cpu_relu(way2_o, batch_size * 192 * out_size * out_size);
    free(way2_o1);
    free(way2_o2);
    free(way2_o3);
    // way3
    double *way3_o = maxpool.cpu_forward(input, batch_size);
    // final
    double *concat_in_final[] = {way1_o, way2_o, way3_o};
    int concat_ch_final[] = {320, 192, 768};
    double *final = cpu_channel_concat(concat_in_final, 3, batch_size, concat_ch_final, out_size, out_size);
    free(way1_o);
    free(way2_o);
    free(way3_o);
    return final;
}

double* InceptionLayer5 :: gpu_forward(double *input, const int batch_size) {
    dim3 grid_conv(8, batch_size);
    dim3 block_conv(32);
    dim3 grid_act(32);
    dim3 block_act(32);
    // way1
    double *way1_o1 = way1_1.basic_forward(grid_conv, block_conv, input, batch_size);
    relu(grid_act, block_act, way1_o1, batch_size * 192 * size * size);
    double *way1_o = way1_2.basic_forward(grid_conv, block_conv, way1_o1, batch_size);
    relu(grid_act, block_act, way1_o, batch_size * 320 * out_size * out_size);
    hipFree(way1_o1);
    // way2
    double *way2_o1 = way2_1.basic_forward(grid_conv, block_conv, input, batch_size);
    relu(grid_act, block_act, way2_o1, batch_size * 192 * size * size);
    double *way2_o2 = way2_2.basic_forward(grid_conv, block_conv, way2_o1, batch_size);
    relu(grid_act, block_act, way2_o2, batch_size * 192 * size * size);
    double *way2_o3 = way2_3.basic_forward(grid_conv, block_conv, way2_o2, batch_size);
    relu(grid_act, block_act, way2_o3, batch_size * 192 * size * size);
    double *way2_o = way2_4.basic_forward(grid_conv, block_conv, way2_o3, batch_size);
    relu(grid_act, block_act, way2_o, batch_size * 192 * out_size * out_size);
    hipFree(way2_o1);
    hipFree(way2_o2);
    hipFree(way2_o3);
    // way3
    double *way3_o = maxpool.basic_forward(grid_conv, block_conv, input, batch_size);
    // final
    double *concat_in_final[] = {way1_o, way2_o, way3_o};
    int concat_ch_final[] = {320, 192, 768};
    double *final = channel_concat(grid_act, block_act, concat_in_final, 3, batch_size, concat_ch_final, out_size, out_size);
    hipFree(way1_o);
    hipFree(way2_o);
    hipFree(way3_o);
    return final;
}

InceptionLayer5 :: ~InceptionLayer5() {}

InceptionLayer6 :: InceptionLayer6(const int in_channels, const int size) : in_channels(in_channels), size(size), way1(in_channels, 320, size, size), way23_1(in_channels, 384, size, size), way2_2(384, 384, size, size, 1, 3, 1, 1, 0, 1), way3_2(384, 384, size, size, 3, 1, 1, 1, 1, 0), way45_1(in_channels, 448, size, size), way45_2(448, 384, size, size, 3, 3, 1, 1, 1, 1), way4_3(384, 384, size, size, 1, 3, 1, 1, 0, 1), way5_3(384, 384, size, size, 3, 1, 1, 1, 1, 0), way6(in_channels, 192, size, size), avgpool(in_channels, size, 3, 1, 1) {
    out_size = size;
    out_channels = 320 + 768 + 768 + 192;
}

int InceptionLayer6 :: get_out_size() const {
    return out_size;
}

int InceptionLayer6 :: get_out_channels() const {
    return out_channels;
}

void InceptionLayer6 :: set_params(struct InceptionLayer6params params) {
    way1.set_params(params.way1_w, params.way1_b);
    way23_1.set_params(params.way23_1_w, params.way23_1_b);
    way2_2.set_params(params.way2_2_w, params.way2_2_b);
    way3_2.set_params(params.way3_2_w, params.way3_2_b);
    way45_1.set_params(params.way45_1_w, params.way45_1_b);
    way45_2.set_params(params.way45_2_w, params.way45_2_b);
    way4_3.set_params(params.way4_3_w, params.way4_3_b);
    way5_3.set_params(params.way5_3_w, params.way5_3_b);
    way6.set_params(params.way6_w, params.way6_b);
}

double* InceptionLayer6 :: cpu_forward(double *input, const int batch_size) {
    // way1
    double *way1_o = way1.cpu_forward(input, batch_size);
    cpu_relu(way1_o, batch_size * 320 * size * size);
    // way2 & way3
    double *way23_o1 = way23_1.cpu_forward(input, batch_size);
    cpu_relu(way23_o1, batch_size * 384 * size * size);
    double *way2_o2 = way2_2.cpu_forward(way23_o1, batch_size);
    cpu_relu(way2_o2, batch_size * 384 * size * size);
    double *way3_o2 = way3_2.cpu_forward(way23_o1, batch_size);
    cpu_relu(way3_o2, batch_size * 384 * size * size);
    double *concat_in1[] = {way2_o2, way3_o2};
    int concat_ch1[] = {384, 384};
    double *way23_o = cpu_channel_concat(concat_in1, 2, batch_size, concat_ch1, size, size);
    free(way23_o1);
    free(way2_o2);
    free(way3_o2);
    // way4 & way5
    double *way45_o1 = way45_1.cpu_forward(input, batch_size);
    cpu_relu(way45_o1, batch_size * 448 * size * size);
    double *way45_o2 = way45_2.cpu_forward(way45_o1, batch_size);
    cpu_relu(way45_o2, batch_size * 384 * size * size);
    double *way4_o3 = way4_3.cpu_forward(way45_o2, batch_size);
    cpu_relu(way4_o3, batch_size * 384 * size * size);
    double *way5_o3 = way5_3.cpu_forward(way45_o2, batch_size);
    cpu_relu(way5_o3, batch_size * 384 * size * size);
    double *concat_in2[] = {way4_o3, way5_o3};
    int concat_ch2[] = {384, 384};
    double *way45_o = cpu_channel_concat(concat_in2, 2, batch_size, concat_ch2, size, size);
    free(way45_o1);
    free(way45_o2);
    free(way4_o3);
    free(way5_o3);
    // way6
    double *way6_o1 = avgpool.cpu_forward(input, batch_size);
    for (int i = 0; i < 100; ++ i) cout << way6_o1[i] << ' ';
    cout << endl;
    double *way6_o = way6.cpu_forward(way6_o1, batch_size);
    cpu_relu(way6_o, batch_size * 192 * size * size);
    free(way6_o1);
    // final
    double *concat_in_final[] = {way1_o, way23_o, way45_o, way6_o};
    int concat_ch_final[] = {320, 768, 768, 192};
    double *final = cpu_channel_concat(concat_in_final, 4, batch_size, concat_ch_final, size, size);
    free(way1_o);
    free(way23_o);
    free(way45_o);
    free(way6_o);
    return final;
}

double* InceptionLayer6 :: gpu_forward(double *input, const int batch_size) {
    dim3 grid_conv(8, batch_size);
    dim3 block_conv(32);
    dim3 grid_act(32);
    dim3 block_act(32);
    // way1
    double *way1_o = way1.basic_forward(grid_conv, block_conv, input, batch_size);
    relu(grid_act, block_act, way1_o, batch_size * 320 * size * size);
    // way2 & way3
    double *way23_o1 = way23_1.basic_forward(grid_conv, block_conv, input, batch_size);
    relu(grid_act, block_act, way23_o1, batch_size * 384 * size * size);
    double *way2_o2 = way2_2.basic_forward(grid_conv, block_conv, way23_o1, batch_size);
    relu(grid_act, block_act, way2_o2, batch_size * 384 * size * size);
    double *way3_o2 = way3_2.basic_forward(grid_conv, block_conv, way23_o1, batch_size);
    relu(grid_act, block_act, way3_o2, batch_size * 384 * size * size);
    double *concat_in1[] = {way2_o2, way3_o2};
    int concat_ch1[] = {384, 384};
    double *way23_o = channel_concat(grid_act, block_act, concat_in1, 2, batch_size, concat_ch1, size, size);
    hipFree(way23_o1);
    hipFree(way2_o2);
    hipFree(way3_o2);
    // way4 & way5
    double *way45_o1 = way45_1.basic_forward(grid_conv, block_conv, input, batch_size);
    relu(grid_act, block_act, way45_o1, batch_size * 448 * size * size);
    double *way45_o2 = way45_2.basic_forward(grid_conv, block_conv, way45_o1, batch_size);
    relu(grid_act, block_act, way45_o2, batch_size * 384 * size * size);
    double *way4_o3 = way4_3.basic_forward(grid_conv, block_conv, way45_o2, batch_size);
    relu(grid_act, block_act, way4_o3, batch_size * 384 * size * size);
    double *way5_o3 = way5_3.basic_forward(grid_conv, block_conv, way45_o2, batch_size);
    relu(grid_act, block_act, way5_o3, batch_size * 384 * size * size);
    double *concat_in2[] = {way4_o3, way5_o3};
    int concat_ch2[] = {384, 384};
    double *way45_o = channel_concat(grid_act, block_act, concat_in2, 2, batch_size, concat_ch2, size, size);
    hipFree(way45_o1);
    hipFree(way45_o2);
    hipFree(way4_o3);
    hipFree(way5_o3);
    // way6
    double *way6_o1 = avgpool.basic_forward(grid_conv, block_conv, input, batch_size);
    double *way6_o1_device = (double *) malloc (sizeof(double) * batch_size * in_channels * size * size);
    hipMemcpy(way6_o1_device, way6_o1, sizeof(double) * batch_size * in_channels * size * size, hipMemcpyDeviceToHost);
    for (int i = 0; i < 100; ++ i) cout << way6_o1_device[i] << ' ';
    cout << endl;
    double *way6_o = way6.basic_forward(grid_conv, block_conv, way6_o1, batch_size);
    relu(grid_act, block_act, way6_o, batch_size * 192 * size * size);
    hipFree(way6_o1);
    // final
    double *concat_in_final[] = {way1_o, way23_o, way45_o, way6_o};
    int concat_ch_final[] = {320, 768, 768, 192};
    double *final = channel_concat(grid_act, block_act, concat_in_final, 4, batch_size, concat_ch_final, size, size);
    hipFree(way1_o);
    hipFree(way23_o);
    hipFree(way45_o);
    hipFree(way6_o);
    return final;
}

InceptionLayer6 :: ~InceptionLayer6() {}