#include "hip/hip_runtime.h"
# include "pooling.h"
# include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <iostream>

using namespace std;

const int batch_size = 8, channels = 2048, size = 17, kernel_size = 3, stride = 1, padding = 1;
MaxpoolingLayer maxpool(channels, size, kernel_size, stride);
MeanpoolingLayer meanpool(channels, size, kernel_size, stride, padding);
const int intput_size = batch_size * channels * size * size;
const int len = (size - kernel_size) / stride + 1;
const int len_mean =  (size + padding * 2 - kernel_size) / stride + 1;
const int output_size_max = channels * len * len;
const int output_size_mean = channels * len_mean * len_mean;

int maxpool_test() {
    double *input;

    input = (double*) malloc (sizeof(double) * batch_size * channels * size * size);
    for (int i = 0; i < batch_size * channels * size * size; ++ i)
        input[i] = (double) (rand() % 32768) / 32768.0;
    
    double *cpu_output = maxpool.cpu_forward(input, batch_size);

    dim3 grid(1, batch_size);
    dim3 block(2);

    double *cuda_input;
    hipMalloc((void **)&cuda_input, sizeof(double) * batch_size * channels * size * size);
    hipMemcpy(cuda_input, input, sizeof(double) * batch_size * channels * size * size, hipMemcpyHostToDevice);
    double *cuda_output = maxpool.basic_forward(grid, block, cuda_input, batch_size);
    double *cuda_output_device;
    cuda_output_device = (double*) malloc (sizeof(double) * batch_size * output_size_max);
    hipMemcpy(cuda_output_device, cuda_output, sizeof(double) * batch_size * output_size_max, hipMemcpyDeviceToHost);

    double max_error = 0.0;
    for (int i = 0; i < batch_size * output_size_max; ++ i) 
        max_error = max(max_error, fabs(cuda_output_device[i] - cpu_output[i]));
    cout << "Max Error = " << max_error << endl;
    if (max_error > 1e-5) cout << "Incorrect.\n";
    else cout << "Correct.\n";

    hipFree(cuda_input);
    hipFree(cuda_output);
    free(input);
    free(cpu_output);
    free(cuda_output_device);
    return 0;    
}

int meanpool_test() {
    double *input;

    input = (double*) malloc (sizeof(double) * batch_size * channels * size * size);
    for (int i = 0; i < batch_size * channels * size * size; ++ i)
        input[i] = (double) (rand() % 32768) / 32768.0;

    double *cpu_output = meanpool.cpu_forward(input, batch_size);


    dim3 grid(8, batch_size);
    dim3 block(32);

    double *cuda_input;
    hipMalloc((void **)&cuda_input, sizeof(double) * batch_size * channels * size * size);
    hipMemcpy(cuda_input, input, sizeof(double) * batch_size * channels * size * size, hipMemcpyHostToDevice);
    double *cuda_output = meanpool.basic_forward(grid, block, cuda_input, batch_size);
    double *cuda_output_device;
    cuda_output_device = (double*) malloc (sizeof(double) * batch_size * output_size_mean);
    hipMemcpy(cuda_output_device, cuda_output, sizeof(double) * batch_size * output_size_mean, hipMemcpyDeviceToHost);

    double max_error = 0.0;
    for (int i = 0; i < batch_size * output_size_mean; ++ i) 
        max_error = max(max_error, fabs(cuda_output_device[i] - cpu_output[i]));
    cout << "Max Error = " << max_error << endl;
    if (max_error > 1e-5) cout << "Incorrect.\n";
    else cout << "Correct.\n";
    
    hipFree(cuda_input);
    hipFree(cuda_output);
    free(input);
    free(cpu_output);
    free(cuda_output_device);
    return 0;    
}

int main(){
    printf("Max pooling test(input size: %d):\n", intput_size);
    maxpool_test();

    printf("Mean pooling test(input size: %d):\n", intput_size);
    meanpool_test();
    return 0;
}