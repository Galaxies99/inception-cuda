#include "hip/hip_runtime.h"
# include "pooling.h"
# include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <iostream>

using namespace std;

const int batch_size = 2048, channels = 2, size = 4, kernel_size = 3, stride = 1, padding = 1;
MaxpoolingLayer maxpool(channels, size, kernel_size, stride);
MeanpoolingLayer meanpool(channels, size, kernel_size, stride, padding);
const int len = size / stride + (size % stride != 0);
const int size_padding = size + padding * 2 - kernel_size + 1;
const int len_mean =  size_padding / stride + (size_padding % stride != 0);
const int output_size_max = channels * len * len;
const int output_size_mean = channels * len_mean * len_mean;
int maxpool_test() {
    float *input;

    input = (float*) malloc (sizeof(float) * batch_size * channels * size * size);
    for (int i = 0; i < batch_size * channels * size * size; ++ i)
        input[i] = (float) (rand() % 32768) / 32768.0;
    
    float *cpu_output = maxpool.cpu_forward(input, batch_size);

    dim3 grid(batch_size);
    dim3 block(channels);

    float *cuda_input;
    hipMalloc((void **)&cuda_input, sizeof(float) * batch_size * channels * size * size);
    hipMemcpy(cuda_input, input, sizeof(float) * batch_size * channels * size * size, hipMemcpyHostToDevice);
    float *cuda_output = maxpool.basic_forward(grid, block, cuda_input, batch_size);
    float *cuda_output_device;
    cuda_output_device = (float*) malloc (sizeof(float) * batch_size * output_size_max);
    hipMemcpy(cuda_output_device, cuda_output, sizeof(float) * batch_size * output_size_max, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < batch_size * output_size_max; ++ i) 
        max_error = max(max_error, fabs(cuda_output_device[i] - cpu_output[i]));
    cout << "Max Error = " << max_error << endl;
    if (max_error > 1e-5) cout << "Incorrect.\n";
    else cout << "Correct.\n";
    return 0;    
}

int meanpool_test() {
    float *input;

    input = (float*) malloc (sizeof(float) * batch_size * channels * size * size);
    for (int i = 0; i < batch_size * channels * size * size; ++ i)
        input[i] = (float) (rand() % 32768) / 32768.0;
    
    // printf("Input:\n");
    // for(int i = 0; i<batch_size;i++){
    //     for(int j = 0; j < channels; j++){
    //         // printf("%d %d\n",i,j);
    //         printf("------ Batch %d Channel %d ------\n", i, j);
    //         for(int x=0;x<size;x++){
    //             for(int y=0;y<size;y++){
    //                 printf("%0.4f ", input[i*channels*size*size+j*size*size+x*size+y]);
    //             }
    //             printf("\n");
    //         }
    //     }
    // }

    float *cpu_output = meanpool.cpu_forward(input, batch_size);

    dim3 grid(batch_size);
    dim3 block(channels);

    float *cuda_input;
    hipMalloc((void **)&cuda_input, sizeof(float) * batch_size * channels * size * size);
    hipMemcpy(cuda_input, input, sizeof(float) * batch_size * channels * size * size, hipMemcpyHostToDevice);
    float *cuda_output = meanpool.basic_forward(grid, block, cuda_input, batch_size);
    float *cuda_output_device;
    cuda_output_device = (float*) malloc (sizeof(float) * batch_size * output_size_mean);
    hipMemcpy(cuda_output_device, cuda_output, sizeof(float) * batch_size * output_size_mean, hipMemcpyDeviceToHost);

    // printf("Output:\n");
    // for(int i = 0; i<batch_size;i++){
    //     for(int j = 0; j < channels; j++){
    //         printf("------ Batch %d Channel %d ------\n", i , j);
    //         for(int x=0;x<len_mean;x++){
    //             for(int y=0;y<len_mean;y++){
    //                 printf("%0.4f ", cpu_output[i*channels*len_mean*len_mean+j*len_mean*len_mean+x*len_mean+y]);
    //             }
    //             printf("\n");
    //         }
    //     }
    // }

    // printf("Output:\n");
    // for(int i = 0; i<batch_size;i++){
    //     for(int j = 0; j < channels; j++){
    //         printf("------ Batch %d Channel %d ------\n", i , j);
    //         for(int x=0;x<len_mean;x++){
    //             for(int y=0;y<len_mean;y++){
    //                 printf("%0.4f ", cuda_output_device[i*channels*len_mean*len_mean+j*len_mean*len_mean+x*len_mean+y]);
    //             }
    //             printf("\n");
    //         }
    //     }
    // }

    float max_error = 0.0;
    for (int i = 0; i < batch_size * output_size_mean; ++ i) 
        max_error = max(max_error, fabs(cuda_output_device[i] - cpu_output[i]));
    cout << "Max Error = " << max_error << endl;
    if (max_error > 1e-5) cout << "Incorrect.\n";
    else cout << "Correct.\n";
    return 0;    
}

int main(){
    printf("Max pooling test:\n");
    maxpool_test();

    printf("Mean pooling test:\n");
    meanpool_test();
    return 0;
}