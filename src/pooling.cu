#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include "pooling.h"


// batch * channel * height * width
__global__ void maxpool_forward(float* bottom_data, const int size, const int kernel_size, float* top_data, float* maxidx)
{
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;

    int i , j, u, v, index, idx;
    float s;
    len = size / kernel_size + (size % kernel_size != 0)
    int index2 = thread_pos * len * len;
    for (i = 0; i < len; ++i)
        for (j = 0; j < len; ++j)
        {
            index = thread_pos * size * size + i * kernel_size * size + j * kernel_size;
            s=-10000.0;
            for (u = 0; u < kernel_size && (u + kernel_size * i) < size; ++u)
                for (v = 0; v < kernel_size && (v + kernel_size * j) < size; ++v)
                    if (*(bottom_data + index + u * size + v) > s){
                        s = *(bottom_data + index + u * size + v);
                        idx = index + u * size + v
                    }
            *(top_data + index2) = s;
            *(maxidx + index2) = idx
            ++index2;
        }
}

__global__ void meanpool_forward(float* bottom_data, const int size, const int kernel_size, float* top_data)
{
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;

    int i , j, u, v, index, idx;
    float s = 0;
    len = size / kernel_size + (size % kernel_size != 0)
    int index2 = thread_pos * len * len;
    for (i = 0; i < len; ++i)
        for (j = 0; j < len; ++j)
        {
            index = thread_pos * size * size + i * kernel_size * size + j * kernel_size;
            for (u = 0; u < kernel_size && (u + kernel_size * i) < size; ++u)
                for (v = 0; v < kernel_size && (v + kernel_size * j) < size; ++v){
                        s += *(bottom_data + index + u * size + v) / (kernel_size * kernel_size);
                }
            *(top_data + index2) = s;
            ++index2;
        }
}