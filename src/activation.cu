#include "hip/hip_runtime.h"
# include "activation.h"


__device__ double activation_relu(double x) {
    return x < 0 ? 0 : x;
}

double activation_relu_cpu(double x) {
    return x < 0 ? 0 : x;
}

__global__ void forward_relu(double *input, double *output, const int size) {
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int begin_idx = 1ll * size * thread_pos / total_threads;
    const int end_idx = 1ll * size * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++ i)
        output[i] = activation_relu(input[i]);
}

double* cpu_relu(double *input, const int size, bool inplace) {
    double *output;
    if (inplace) output = input;
    else output = (double*) malloc (sizeof(double) * size);
    for (int i = 0; i < size; ++ i)
        output[i] = activation_relu_cpu(input[i]);
    return output;
}

double* relu(dim3 grid, dim3 block, double *input, const int size, bool inplace) {
    double *output;
    if (inplace) output = input;
    else hipMalloc((void **)&output, sizeof(double) * size);
    forward_relu <<<grid, block>>> (input, output, size);
    return output;
}