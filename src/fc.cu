#include "hip/hip_runtime.h"
# include "fc.h"

// Construction function of fully connected layer.
FullyConnectedLayer :: FullyConnectedLayer(int _in_features, int _out_features) {
    in_features = _in_features;
    out_features = _out_features;

    weight_N = in_features * out_features;
    bias_N = out_features;
    output_N = _out_features;

    hipMalloc(&weight, sizeof(double) * weight_N);
    hipMalloc(&bias, sizeof(double) * bias_N);
    set_params();
}

// Destruction function of fully connected layer.
FullyConnectedLayer :: ~FullyConnectedLayer() {
    hipFree(weight);
    hipFree(bias);
    if (h_weight != NULL) free(h_weight);
    if (h_bias != NULL) free(h_bias);
}

// FC forward (cpu)
void fc_forward_cpu(double *input, double *output, double *weight, double *bias, const int batch_size, const int in_features, const int out_features) {
    const int input_N = in_features;
    const int output_N = out_features;
    for (int b = 0; b < batch_size; ++b) {
        for (int row = 0; row < out_features; ++row) {
            output[b * output_N + row] = bias[row];
            for (int col = 0; col < in_features; ++col) {
                output[b * output_N + row] += weight[row * in_features + col] * input[b * input_N + col];
            }
        }
    }
}

double* FullyConnectedLayer :: cpu_forward(double *input, const int batch_size) {
    double *output;
    output = (double *) malloc (sizeof(double) * batch_size * output_N);
    fc_forward_cpu(input, output, h_weight, h_bias, batch_size, in_features, out_features);
    return output;
}

// FC forward (weight) (basic, no optimization)
__global__ void fc_basic_weight_forward(double *input, double *output, double *weight, const int batch_size, const int in_features, const int out_features) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int input_N = in_features;
    const int output_N = out_features;
    const int total_N = in_features * out_features;
    const int begin_idx = total_N * thread_pos / total_threads;
    const int end_idx = total_N * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++i) {
        const int col = i % in_features;
        const int row = (i / in_features) % out_features;
        atomicAdd(&output[batch_id * output_N + row], weight[row * in_features + col] * input[batch_id * input_N + col]);
    }
}

// FC forward (bias) (basic, no optimization)
__global__ void fc_basic_bias_forward(double *input, double *output, double *bias, const int batch_size, const int in_features, const int out_features) {
    const int batch_id = blockIdx.y;
    const int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int output_N = out_features;
    const int begin_idx = output_N * thread_pos / total_threads;
    const int end_idx = output_N * (thread_pos + 1) / total_threads;
    for (int i = begin_idx; i < end_idx; ++i) {
        const int idx = i % out_features;
        output[batch_id * output_N + idx] = input[batch_id * output_N + idx] + bias[idx];
    }
}

double* FullyConnectedLayer :: basic_forward(dim3 grid, dim3 block, double *input, const int batch_size) {
    double *output;
    hipMalloc((void **)&output, sizeof(double) * batch_size * output_N);
    hipMemset(output, 0, sizeof(double) * batch_size * output_N);
    fc_basic_weight_forward <<<grid, block>>> (input, output, weight, batch_size, in_features, out_features);
    fc_basic_bias_forward <<<grid, block>>> (output, output, bias, batch_size, in_features, out_features);
    hipDeviceSynchronize();
    return output;
}

void FullyConnectedLayer :: set_params(double *_h_weight, double *_h_bias) {
    if (_h_weight == NULL) {
        h_weight = (double*) malloc (sizeof(double) * weight_N);
        for (int i = 0; i < weight_N; ++i)
            h_weight[i] = init_rand();
    } else {
        if (h_weight != NULL) free(h_weight);
        h_weight = _h_weight;
    }
    if (_h_bias == NULL) {
        h_bias = (double*) malloc (sizeof(double) * bias_N);
        for (int i = 0; i < bias_N; ++i)
            h_bias[i] = init_rand();
    } else {
        if (h_bias != NULL) free(h_bias);
        h_bias = _h_bias;
    }
    hipMemcpy(weight, h_weight, sizeof(double) * weight_N, hipMemcpyHostToDevice);
    hipMemcpy(bias, h_bias, sizeof(double) * bias_N, hipMemcpyHostToDevice);
}

void fc_cudnn_forward(
    hipdnnHandle_t& handle,
    double *input,
    double *output,
    double *weight,
    double *bias,
    const int batch_size,
    const int in_channels,
    const int out_channels
) {
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            in_channels,
            1,
            1
        )
    );
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(
        hipdnnSetFilter4dDescriptor(
            kernel_descriptor,
            HIPDNN_DATA_DOUBLE,
            HIPDNN_TENSOR_NCHW,
            out_channels,
            in_channels,
            1,
            1
        )
    );
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(
        hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            0,
            0,
            1,
            1,
            1,
            1,
            HIPDNN_CROSS_CORRELATION,
            HIPDNN_DATA_DOUBLE
        )
    );
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            out_channels,
            1,
            1
        )
    );
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    hipdnnConvolutionFwdAlgoPerf_t convolution_algorithm_perf[4];
    int returned_cnt;
    checkCUDNN(
        cudnnGetConvolutionForwardAlgorithm_v7(
            handle,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            4,
            &returned_cnt,
            convolution_algorithm_perf
        )
    );
    bool found_algo = false;
    for (int n = 0; n < returned_cnt; ++ n) {
        if (convolution_algorithm_perf[n].status == HIPDNN_STATUS_SUCCESS && 
            convolution_algorithm_perf[n].algo != HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED) {
            convolution_algorithm = convolution_algorithm_perf[n].algo;
            found_algo = true;
            break;
        }
    }
    if (! found_algo) {
        std :: cerr << "No convolution algorithm is found." << std :: endl;
        std :: exit(EXIT_FAILURE);
    }
    size_t workspace_bytes;
    checkCUDNN(
        hipdnnGetConvolutionForwardWorkspaceSize(
            handle,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            convolution_algorithm,
            &workspace_bytes
        )
    );
    if (workspace_bytes == 0) workspace_bytes = (size_t)(1 << 23);
    void *workspace = NULL;
    hipMalloc(&workspace, workspace_bytes);
    const double alpha = 1.0, beta = 0.0;
    const size_t output_bytes = sizeof(double) * batch_size * out_channels;
    checkCUDNN(
        hipdnnConvolutionForward(
            handle,
            &alpha,
            input_descriptor,
            input,
            kernel_descriptor,
            weight,
            convolution_descriptor,
            convolution_algorithm,
            workspace,
            output_bytes,
            &beta,
            output_descriptor,
            output
        )
    );

    hipFree(workspace);

    if (bias != NULL) {
        hipdnnTensorDescriptor_t bias_descriptor;
        checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCUDNN(
            hipdnnSetTensor4dDescriptor(
                bias_descriptor,
                HIPDNN_TENSOR_NCHW,
                HIPDNN_DATA_DOUBLE,
                1,
                out_channels,
                1,
                1
            )
        );
        checkCUDNN(
            hipdnnAddTensor(
                handle,
                &alpha,
                bias_descriptor,
                bias,
                &alpha,
                output_descriptor,
                output
            )
        );
        hipdnnDestroyTensorDescriptor(bias_descriptor);
    }

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
}

double* FullyConnectedLayer :: cudnn_forward(hipdnnHandle_t &handle, double *input, const int batch_size) {
    double *output;
    hipMalloc((void **)&output, sizeof(double) * batch_size * output_N);
    hipMemset(output, 0, sizeof(double) * batch_size * output_N);
    fc_cudnn_forward(handle, input, output, weight, bias, batch_size, in_features, out_features);
    return output;
}
