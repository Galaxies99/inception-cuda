# include "inception.h"


void conv_forward(
    hipdnnHandle_t& handle,
    double *input,
    double *output,
    double *weight,
    double *bias,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int size_r,
    const int size_c,
    const int kernel_size_r,
    const int kernel_size_c,
    const int stride_r,
    const int stride_c,
    const int padding_r,
    const int padding_c,
    const bool with_relu
) {
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            in_channels,
            size_r,
            size_c
        )
    );

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(
        hipdnnSetFilter4dDescriptor(
            kernel_descriptor,
            HIPDNN_DATA_DOUBLE,
            HIPDNN_TENSOR_NCHW,
            out_channels,
            in_channels,
            kernel_size_r,
            kernel_size_c
        )
    );

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(
        hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            padding_r,
            padding_c,
            stride_r,
            stride_c,
            1,
            1,
            HIPDNN_CROSS_CORRELATION,
            HIPDNN_DATA_DOUBLE
        )
    );

    const int out_size_r = (size_r - kernel_size_r + 2 * padding_r) / stride_r + 1;
    const int out_size_c = (size_c - kernel_size_c + 2 * padding_c) / stride_c + 1;

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            out_channels,
            out_size_r,
            out_size_c
        )
    );

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(
            handle,
			input_descriptor,
			kernel_descriptor,
			convolution_descriptor,
			output_descriptor,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
            0,
			&convolution_algorithm
        )
    );

    size_t workspace_bytes;
    checkCUDNN(
        hipdnnGetConvolutionForwardWorkspaceSize(
            handle,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            convolution_algorithm,
            &workspace_bytes
        )
    );

    if (workspace_bytes == 0) workspace_bytes = (size_t)(1 << 23);
    assert (workspace_bytes > 0);
    void *workspace = NULL;
    hipMalloc(&workspace, workspace_bytes);

    const double alpha = 1.0, beta = 0.0;
    const size_t output_bytes = sizeof(double) * batch_size * out_channels * out_size_r * out_size_c;

    checkCUDNN(
        hipdnnConvolutionForward(
            handle,
            &alpha,
            input_descriptor,
            input,
            kernel_descriptor,
            kernel,
            convolution_descriptor,
            convolution_algorithm,
            workspace,
            output_bytes,
            &beta,
            output_descriptor,
            output
        )
    );

    if (bias != NULL) {
        hipdnnTensorDescriptor_t bias_descriptor;
        checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCUDNN(
            hipdnnSetTensor4dDescriptor(
                bias_descriptor,
                HIPDNN_TENSOR_NCHW,
                HIPDNN_DATA_DOUBLE,
                1,
                out_channels,
                1,
                1
            )
        );
        checkCUDNN(
            hipdnnAddTensor(
                handle,
                &alpha,
                bias_descriptor,
                bias,
                &beta,
                output_descriptor,
                output
            )
        );
        hipdnnDestroyTensorDescriptor(bias_descriptor);
    }

    if (with_relu) {
        hipdnnActivationDescriptor_t activation_descriptor;
        checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
        checkCUDNN(
            hipdnnSetActivationDescriptor(
                activation_descriptor,
                HIPDNN_ACTIVATION_RELU,
                HIPDNN_PROPAGATE_NAN,
                0
            )
        );
        checkCUDNN(
            hipdnnActivationForward(
                handle,
                activation_descriptor,
                &alpha,
                output_descriptor,
                output,
                &beta,
                output_descriptor,
                output
            )
        );
        hipdnnDestroyActivationDescriptor(activation_descriptor);
    }

    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
}