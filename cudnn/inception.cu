# include "inception.h"

void conv_forward_layer(
    hipdnnHandle_t& handle,
    double *input,
    double *output,
    double *weight,
    double *bias,
    const int batch_size,
    const int in_channels,
    const int out_channels,
    const int size_r,
    const int size_c,
    const int kernel_size_r,
    const int kernel_size_c,
    const int stride_r,
    const int stride_c,
    const int padding_r,
    const int padding_c
) {
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(
        hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_DOUBLE,
            batch_size,
            in_channels,
            size_r,
            size_c
        )
    );
}